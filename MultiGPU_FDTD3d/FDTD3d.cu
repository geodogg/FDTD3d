#include "hip/hip_runtime.h"
// C LIBRARY INCLUDES
#include <cstdio>
#include <cassert>  // useful for debugging
#include <cstdlib>
#include <ctime>
using namespace std;

// CUDA LIBRARY INCLUDES
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>
// #include <helper_functions.h>
// #include <hip/hip_cooperative_groups.h>

// EXTRA
#include "functions.h"

// GPU ERROR CHECKING MACROS
#define gpuErrchk(ans){ gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char * file, int line, bool abort = true){
  if (code != hipSuccess){
    fprintf(stderr, "gpuAssert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}
#define printline(ans) { fprintf(outfile, "file: %s line: %d\n - ", __FILE__, __LINE__); fprintf(outfile, ans); }

// DEFINE CONSTANTS DATA
#define dimx 50
#define dimy 50
#define dimz 50
#define radius 4
#define timesteps 5
#define outerDimx 384
#define outerDimy 384
#define outerDimz 384
#define volumeSize 56623104
#define lowerBound 0.0f
#define upperBound 1.0f
#define padding 28
#define paddedVolumeSize 56623132
#define size (sizeof(float))
// struct hipPitchedPtr{
//   size_t = pitch, xsize, ysize;
//   void * = ptr;
// }
// struct hipExtent{
//   size_t = width, height, depth;
// }
// KERNELS
// #define RADIUS 4
// __constant__ float stencil[RADIUS + 1];
// __global__ void FiniteDifferencesKernel(float *bufferDst, float *bufferSrc)
// {
//     bool validr = true;
//     bool validw = true;
//     const int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
//     const int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
//     const int ltidx = threadIdx.x;
//     const int ltidy = threadIdx.y;
//     const int workx = blockDim.x;
//     const int worky = blockDim.y;
//     // Handle to thread block group
//     cg::thread_block cta = cg::this_thread_block();
//     __shared__ float tile[k_blockDimMaxY + 2 * RADIUS][k_blockDimX + 2 * RADIUS];
//
//     const int stride_y = dimx + 2 * RADIUS;
//     const int stride_z = stride_y * (dimy + 2 * RADIUS);
//
//     int inputIndex  = 0;
//     int outputIndex = 0;
//
//     // Advance inputIndex to start of inner volume
//     inputIndex += RADIUS * stride_y + RADIUS;
//
//     // Advance inputIndex to target element
//     inputIndex += gtidy * stride_y + gtidx;
//
//     float infront[RADIUS];
//     float behind[RADIUS];
//     float current;
//
//     const int tx = ltidx + RADIUS;
//     const int ty = ltidy + RADIUS;
//
//     // Check in bounds
//     if ((gtidx >= dimx + RADIUS) || (gtidy >= dimy + RADIUS))
//         validr = false;
//
//     if ((gtidx >= dimx) || (gtidy >= dimy))
//         validw = false;
//
//     // Preload the "infront" and "behind" data
//     for (int i = RADIUS - 2 ; i >= 0 ; i--)
//     {
//         if (validr)
//             behind[i] = bufferSrc[inputIndex];
//
//         inputIndex += stride_z;
//     }
//
//     if (validr)
//         current = bufferSrc[inputIndex];
//
//     outputIndex = inputIndex;
//     inputIndex += stride_z;
//
//     for (int i = 0 ; i < RADIUS ; i++)
//     {
//         if (validr)
//             infront[i] = bufferSrc[inputIndex];
//
//         inputIndex += stride_z;
//     }
//
//     // Step through the xy-planes
//     #pragma unroll 9
//
//     for (int iz = 0 ; iz < dimz ; iz++)
//     {
//         // Advance the slice (move the thread-front)
//         for (int i = RADIUS - 1 ; i > 0 ; i--)
//             behind[i] = behind[i - 1];
//
//         behind[0] = current;
//         current = infront[0];
//         #pragma unroll 4
//
//         for (int i = 0 ; i < RADIUS - 1 ; i++)
//             infront[i] = infront[i + 1];
//
//         if (validr)
//             infront[RADIUS - 1] = bufferSrc[inputIndex];
//
//         inputIndex  += stride_z;
//         outputIndex += stride_z;
//         cg::sync(cta);
//
//         // Note that for the work items on the boundary of the problem, the
//         // supplied index when reading the halo (below) may wrap to the
//         // previous/next row or even the previous/next xy-plane. This is
//         // acceptable since a) we disable the output write for these work
//         // items and b) there is at least one xy-plane before/after the
//         // current plane, so the access will be within bounds.
//
//         // Update the data slice in the local tile
//         // Halo above & below
//         if (ltidy < RADIUS)
//         {
//             tile[ltidy][tx]                  = bufferSrc[outputIndex - RADIUS * stride_y];
//             tile[ltidy + worky + RADIUS][tx] = bufferSrc[outputIndex + worky * stride_y];
//         }
//
//         // Halo left & right
//         if (ltidx < RADIUS)
//         {
//             tile[ty][ltidx]                  = bufferSrc[outputIndex - RADIUS];
//             tile[ty][ltidx + workx + RADIUS] = bufferSrc[outputIndex + workx];
//         }
//
//         tile[ty][tx] = current;
//         cg::sync(cta);
//
//         // Compute the output value
//         float value = stencil[0] * current;
//         #pragma unroll 4
//
//         for (int i = 1 ; i <= RADIUS ; i++)
//         {
//             value += stencil[i] * (infront[i-1] + behind[i-1] + tile[ty - i][tx] + tile[ty + i][tx] + tile[ty][tx - i] + tile[ty][tx + i]);
//         }
//
//         // Store the output value
//         if (validw)
//             bufferDst[outputIndex] = value;
//     }
// }

// Initialize Unified Memory
__device__ __managed__ float input[dimx * dimy * dimz + 8];

int main(int argc, char * argv[]){
    printf("Running program: %s\n", argv[0]);

    // outfile for debugging
    FILE * outfile;
    outfile = fopen("debug.txt", "w");
    if (outfile == NULL){
      printf(".....there is an error opening debug file....\n");
      return 0;
    }

    // allocate 3D device memory
    hipPitchedPtr PDP;  // pitchedDevPtr
    hipExtent volume_bytes = make_hipExtent(size * dimx, size * dimy, size * dimz);
    hipMalloc3D(&PDP, volume_bytes);

    printf ("width: %d\nheight: %d\ndepth: %d\n", volume_bytes.width, volume_bytes.height, volume_bytes.depth);
    printf ("pitch: %d\npointer: %p\nxsize: %d\nysize: %d\n", PDP.pitch, PDP.ptr, PDP.xsize, PDP.ysize);

    printline("good\n")

    // copy the 3D device memory to Unified Memory
    gpuErrchk(hipMemcpy(input, PDP.ptr, PDP.pitch * PDP.xsize * PDP.ysize, hipMemcpyDefault))


    for(int i = 0; i < 100; i++) {

      printline("good\n")
      fprintf(outfile, "input[%d] = %f\n", i, input[i]);

      printline("good\n")
      //zfprintf(outfile, "PDP[%d] = %f\n", i, *(PDP.ptr));
    }



    // // Get the memory size of the target device and save in memsize
    // getTargetDeviceGlobalMemSize(&memsize, argc, argv);
    // memsize /= 2;
    // printf("Memory size: %d\n", memsize);
    //
    // printf(" generateRandomData\n\n");
    // generateRandomData(input, outerDimx, outerDimy, outerDimz, lowerBound, upperBound);
    // printf("FDTD on %d x %d x %d volume with symmetric filter radius %d for %d timesteps...\n\n", dimx, dimy, dimz, radius, timesteps);
    //
    // gpuErrchk(hipMemcpy(buffer_in + padding, input, volumeSize * sizeof(float), hipMemcpyDefault));
    // gpuErrchk(hipMemcpy(buffer_out + padding, input, volumeSize * sizeof(float), hipMemcpyDefault));
    //
    // // Set up block and grid
    // dim3 dimBlock;
    // dim3 dimGrid;
    // dimBlock.x = 32;
    // dimBlock.y = 16;
    // dimGrid.x = 12;
    // dimGrid.y = 24;
    // // Execute the FDTD
    // float *bufferSrc = buffer_in + padding;
    // float *bufferDst = buffer_out + padding;
    // printf(" GPU FDTD loop\n");
    //
    // clock_t tic = clock();  // start clocking
    //
    // for (int it = 0 ; it < timesteps ; it++){
    //
    //   printf("\tt = %d ", it);
    //
    //   // Launch the kernel
    //   printf("launch kernel\n");
    //   FiniteDifferencesKernel<<<dimGrid, dimBlock>>>(bufferDst, bufferSrc);
    //
    //   float *tmp = bufferDst;
    //   bufferDst = bufferSrc;
    //   bufferSrc = tmp;
    // }
    //
    // clock_t toc = clock() - tic;
    // float elapsed_time = ((float)toc) / CLOCKS_PER_SEC;   // finish clocking
    // printf("Vector addition on the DEVICE\nElapsed time: %f (sec)\n", elapsed_time);
    //
    // for(int i = 0; i < paddedVolumeSize; i+=(141376))
    //   fprintf(outfile, "input[%d] = %f\n", i, bufferDst[i]);

    fclose(outfile);
    return 0;
}
