// C LIBRARY INCLUDES
#include <cstdio>
#include <cassert>  // useful for debugging
#include <cstdlib>
using namespace std;

// CUDA LIBRARY INCLUDES
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

// EXTRA
#include "functions.h"

bool getTargetDeviceGlobalMemSize(memsize_t *result, int argc, char **argv)
{
    int               deviceCount  = 0;
    int               targetDevice = 0;
    size_t            memsize      = 0;

    // Get the number of CUDA enabled GPU devices
    printf(" hipGetDeviceCount\n");
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    // Select target device (device 0 by default)
    targetDevice = findCudaDevice(argc, (const char **)argv);

    // Query target device for maximum memory allocation
    printf(" hipGetDeviceProperties\n");
    struct hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

    memsize = deviceProp.totalGlobalMem;

    // Save the result
    *result = (memsize_t)memsize;
    return true;
}
